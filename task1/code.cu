#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define Tile_size 2


int numARows;   
int numAColumns;  
int numBRows;   
int numBColumns;  
int numCRows;  
int numCColumns; 


//Kernel for shared memory/ Tiled execution
__global__ void matrixMultiplyShared(float* A, float* B, float* C,
	int numARows, int numAColumns,
	int numBRows, int numBColumns,
	int numCRows, int numCColumns)
{
	// Tile size to store elements in shared memory
	__shared__ float sA[Tile_size][Tile_size];
	__shared__ float sB[Tile_size][Tile_size];

	//To generate ids of threads.
	int Row = blockDim.y * blockIdx.y + threadIdx.y;
	int Col = blockDim.x * blockIdx.x + threadIdx.x;
	float Cvalue = 0.0;
	sA[threadIdx.y][threadIdx.x] = 0.0;
	sB[threadIdx.y][threadIdx.x] = 0.0;

	for (int k = 0; k < (((numAColumns - 1) / Tile_size) + 1); k++)
	{
		//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
		if ((Row < numARows) && (threadIdx.x + (k * Tile_size)) < numAColumns)
		{
			sA[threadIdx.y][threadIdx.x] = A[(Row * numAColumns) + threadIdx.x + (k * Tile_size)];
		}
		else
		{
			sA[threadIdx.y][threadIdx.x] = 0.0;
		}
		//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
		if (Col < numBColumns && (threadIdx.y + k * Tile_size) < numBRows)
		{
			sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * Tile_size) * numBColumns + Col];
		}
		else
		{
			sB[threadIdx.y][threadIdx.x] = 0.0;
		}
		__syncthreads();

		//Multiplying Elements present in tile
		for (int j = 0; j < Tile_size; ++j)
		{
			Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
		}
	}
	//Saving Final result into Matrix C
	if (Row < numCRows && Col < numCColumns)
	{
		C[Row * numCColumns + Col] = Cvalue;
	}
}

//Function To print the Matrix
void Print_Mat(int Row, int Col, float* Mat)
{
	for (int i = 0; i < Row * Col; i++)
	{
		printf("%f  ", *(Mat + i));

		if ((i % Col) == 0)
		{
			printf("\n");
		}
	}
}

//Normal CPU Matrix Multiplication
void matMultiplyOnHost(float* A, float* B, float* C, int numARows,
	int numAColumns, int numBRows, int numBColumns,
	int numCRows, int numCColumns)
{
	for (int i = 0; i < numARows; i++)
	{
		for (int j = 0; j < numAColumns; j++)
		{
			C[i * numCColumns + j] = 0.0;
			for (int k = 0; k < numCColumns; k++)
			{
				C[i * numCColumns + j] += A[i * numAColumns + k] * B[k * numBColumns + j];
			}
		}
	}
	return;
}

int main(int argc, char** argv) {
	float* hostA;
	float* hostB;
	float* hostC;
	float* hostComputedC;
	float* deviceA;
	float* deviceB;
	float* deviceC;

	printf("\nPlease Enter Rows and Columns of A:");
	scanf("%d %d", &numARows, &numAColumns);

	printf("\nPlease Enter Rows and Columns of B:");
	scanf("%d %d", &numBRows, &numBColumns);

	hostA = (float*)malloc(sizeof(float) * numARows * numAColumns);
	hostB = (float*)malloc(sizeof(float) * numBRows * numBColumns);

	//Matrix Initialization
	for (int i = 0; i < numARows * numAColumns; i++)
	{
		hostA[i] = 1.0;
	}
	for (int i = 0; i < numBRows * numBColumns; i++)
	{
		hostB[i] = 1.0;
	}

	printf("\nMatrix A Values:\n");
	Print_Mat(numARows, numAColumns, hostA);

	printf("\n\nMatrix B Values:\n");
	Print_Mat(numBRows, numBColumns, hostB);



	numCRows = numARows;
	numCColumns = numBColumns;

	hostC = (float*)malloc(sizeof(float) * numCRows * numCColumns);
	hostComputedC = (float*)malloc(sizeof(float) * numCRows * numCColumns);

	// Allocating GPU memory
	hipMalloc((void**)&deviceA, sizeof(float) * numARows * numAColumns);
	hipMalloc((void**)&deviceB, sizeof(float) * numBRows * numBColumns);
	hipMalloc((void**)&deviceC, sizeof(float) * numCRows * numCColumns);

	// Copy memory to the GPU
	hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);

	// Initialize the grid and block dimensions
	dim3 dimGrid((numCColumns / Tile_size) + 1, (numCRows / Tile_size) + 1, 1);//Number of Blocks required
	dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block

	//Launch the GPU Kernel here
	matrixMultiplyShared << <dimGrid, dimBlock >> > (deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	hipDeviceSynchronize();//To synchronize the device

	// Copy the results in GPU memory back to the CPU
	hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);

	printf("\nMatrix C From Device\n");
	Print_Mat(numCRows, numCColumns, hostC);

	matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

	printf("\nMatrix C From Host\n");
	Print_Mat(numCRows, numCColumns, hostComputedC);

	for (int i = 0; i < numCColumns * numCRows; i++)//Compare both the result matrices 1. MatrixMultiplyonHost 2. MatrixMultiplyonDevice
	{
		if (hostComputedC[i] != hostC[i])
		{
			printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
			break;
		}
	}

	printf("\n Number of Blocks Created:%d \n", ((numCColumns / Tile_size) + 1) * ((numCColumns / Tile_size) + 1));
	printf("\n Number of Threads Per Block: %d \n", (Tile_size * Tile_size));

	// Free the GPU memory
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	//Free the Pointer Memory
	free(hostA);
	free(hostB);
	free(hostC);
	free(hostComputedC);

	return 0;
}
